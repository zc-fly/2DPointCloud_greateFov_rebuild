#include "hip/hip_runtime.h"
#include "ImageRender.cuh"

__global__ static void GaussKernal(uchar* Image, float* Table, int ROISize, int N, int Width)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int x = Table[3 * i];
	int y = Table[3 * i + 1];
	int c = 0.03*Table[3 * i + 2];

	for (int p = 0; p < ROISize; p++) {
		for (int q = 0; q < ROISize; q++) {
			if (i < N) {
				int x_current = x + p - ((ROISize - 1) / 2);
				int y_current = y + q - ((ROISize - 1) / 2);
				Image[x_current*Width + y_current] = 200*exp(-(powf((x_current - x), 2) + powf((y_current - y), 2)) / (2 * c*c));
			}
		}
	}
}

ImageRender::ImageRender(Result table, int imgRawSize, int renderdPixelSize, int gaussianKernalSize) {
		r_table = table;
		r_imgRawSize = imgRawSize;
		r_renderdPixelSize = renderdPixelSize;
		r_gaussianKernalSize = gaussianKernalSize;
	}

void ImageRender::Render() {
	int MaxX = 0, MaxY = 0;
	for (int i = 0; i < r_table.Num; i++) {
		r_table.pointData[M_nano*i] = round(r_table.pointData[M_nano*i] * r_imgRawSize / r_renderdPixelSize) + floor(r_gaussianKernalSize / 2 + 1);
		r_table.pointData[M_nano*i + 1] = round(r_table.pointData[M_nano*i + 1] * r_imgRawSize / r_renderdPixelSize) + floor(r_gaussianKernalSize / 2 + 1);
		r_table.pointData[M_nano*i + 2] = r_table.pointData[M_nano*i + 2] * r_imgRawSize / r_renderdPixelSize;//ת��Ϊ���ظ���
		if (r_table.pointData[M_nano*i] > MaxX) MaxX = r_table.pointData[M_nano*i];
		if (r_table.pointData[M_nano*i + 1] > MaxY) MaxY = r_table.pointData[M_nano*i + 1];
	}

	cv::Mat renderImage = cv::Mat::zeros(MaxX + 10, MaxY + 10, CV_8UC1);
	
	size_t memSize_Image = renderImage.cols * renderImage.rows * sizeof(uchar);//һ���ֽڣ�8λ��256����ӦCV_8UC1
	size_t memSize_Table = M_nano * r_table.Num * sizeof(float);
	uchar* Image_cuda = NULL;
	float* pointTable_cuda = NULL;
	hipMalloc((void**)&Image_cuda, memSize_Image);
	hipMalloc((void**)&pointTable_cuda, memSize_Table);
	hipMemcpy(Image_cuda, renderImage.data ,memSize_Image, hipMemcpyHostToDevice);
	hipMemcpy(pointTable_cuda, r_table.pointData, memSize_Table, hipMemcpyHostToDevice);
	int Num = r_table.Num / 1024 + 1;
	GaussKernal << <Num, 1024 >> > (Image_cuda, pointTable_cuda, r_gaussianKernalSize, r_table.Num, renderImage.cols);
	hipMemcpy(renderImage.data, Image_cuda, memSize_Image, hipMemcpyDeviceToHost);
	hipFree(Image_cuda);
	hipFree(pointTable_cuda);

	cv::namedWindow("RenderdImage", CV_WINDOW_NORMAL);
	cv::imshow("RenderdImage",renderImage);
	cv::waitKey(0);
}