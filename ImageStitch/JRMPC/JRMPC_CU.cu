#include "hip/hip_runtime.h"
#include "JRMPC_CU.cuh"

using namespace std;
using namespace Eigen;


__global__ static void MatrixEularDis(float *P, float *C, float *A, int N, int K)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	for (int j = 0; j < K; j++) {
		if (i < N) {
			A[j*N + i] = abs(P[i] + C[j] - 2 * A[j*N+i]);
		}
	}

}


void JRMPC::setInputPoint(struct Result *Data_1, struct Result *Data_2)
{
	PointData_1 = *Data_1;
	PointData_2 = *Data_2;
}

void JRMPC::edgeArea_extraction(int Edge, int Flag)
{
	int Num_1 = 0;
	int Num_2 = 0;
	float Max_1 = 0;
	float Min_2 = INT_MAX;
	float(*point_1)[M_Nano] = (float(*)[M_Nano]) PointData_1.pointData;
	float(*point_2)[M_Nano] = (float(*)[M_Nano]) PointData_2.pointData;

	if (Flag == 1) {
		for (int i = 0; i < PointData_1.Num; i++)//find the Max or Min Value
			if (point_1[i][0] > Max_1) Max_1 = point_1[i][0];
		for (int i = 0; i < PointData_2.Num; i++)
			if (point_2[i][0] < Min_2) Min_2 = point_2[i][0];
		for (int i = 0; i < PointData_1.Num; i++) {//Count the edgeArea points nums
			if (point_1[i][0] > Max_1 - Edge) Num_1++;
		}
		for (int i = 0; i < PointData_2.Num; i++) {//Count the edgeArea points nums
			if (point_2[i][0] < Min_2 + Edge) Num_2++;
		}
		int i_1 = 0;
		float *pointEdge_1 = new float[Num_1*M_Nano];
		for (int i = 0; i < PointData_1.Num; i++) {//Get the edgeArea points
			if (point_1[i][0] > Max_1 - Edge) {
				for (int j = 0; j < M_Nano; j++) {
					pointEdge_1[i_1] = point_1[i][j];
					i_1++;
				}
			}
		}
		delete[] PointData_1.pointData;

		int i_2 = 0;
		float *pointEdge_2 = new float[Num_2*M_Nano];
		for (int i = 0; i < PointData_2.Num; i++) {//Get the edgeArea points
			if (point_2[i][0] < Min_2 + Edge) {
				for (int j = 0; j < M_Nano; j++) {
					pointEdge_2[i_2] = point_2[i][j];
					i_2++;
				}
			}
		}
		delete[] PointData_2.pointData;

		PointData_1.Num = Num_1;
		PointData_1.pointData = pointEdge_1;
		PointData_2.Num = Num_2;
		PointData_2.pointData = pointEdge_2;

	}
	else if (Flag == 2) {
		for (int i = 0; i < PointData_1.Num; i++)//find the Max or Min Value
			if (point_1[i][1] > Max_1) Max_1 = point_1[i][1];
		for (int i = 0; i < PointData_2.Num; i++)
			if (point_2[i][1] < Min_2) Min_2 = point_2[i][1];
		for (int i = 0; i < PointData_1.Num; i++) {//Count the edgeArea points nums
			if (point_1[i][1] > Max_1 - Edge) Num_1++;
		}
		for (int i = 0; i < PointData_2.Num; i++) {//Count the edgeArea points nums
			if (point_2[i][1] < Min_2 + Edge) Num_2++;
		}
		int i_1 = 0;
		float *pointEdge_1 = new float[Num_1*M_Nano];
		for (int i = 0; i < PointData_1.Num; i++) {//Get the edgeArea points
			if (point_1[i][1] > Max_1 - Edge) {
				for (int j = 0; j < M_Nano; j++) {
					pointEdge_1[i_1] = point_1[i][j];
					i_1++;
				}
			}
		}
		delete[] PointData_1.pointData;

		int i_2 = 0;
		float *pointEdge_2 = new float[Num_2*M_Nano];
		for (int i = 0; i < PointData_2.Num; i++) {//Get the edgeArea points
			if (point_2[i][1] < Min_2 + Edge) {
				for (int j = 0; j < M_Nano; j++) {
					pointEdge_2[i_2] = point_2[i][j];
					i_2++;
				}
			}
		}
		delete[] PointData_2.pointData;

		PointData_1.Num = Num_1;
		PointData_1.pointData = pointEdge_1;
		PointData_2.Num = Num_2;
		PointData_2.pointData = pointEdge_2;
	}
	else {
		cout << "Set a correct Flag Value";
		exit(0);
	}

};

/*
void JRMPC::preProcess()
{

};*/

MatrixXf JRMPC::jrmpcMethod()
{

	//Setting Paraments for JRMPC
	const int K = this->K;//�������ĵ�����
	const int maxNumIter = this->maxNumIter;//��������
	double epsilon = pow(10.0,-5);
	float gamma = 0.0500;//(1 / K)
	Matrix<float, K, 1> pk;//K���㼯Ȩ��+�����㼯Ȩ��
	MatrixXf moveDistanceXY(1, 2);
	//*******pk = 0*pk.array() + (1 / (K*(gamma + 1)));
	pk = 0 * pk.array() + 0.003200;

	//vector<pair<float, float>> T(2);//λ����
	Matrix<float, 2, 2> T;
	T.fill(0);
	//MatrixXf T = MatrixXf::Zero(2,2);//λ����

	//Create Data Matrix
	Matrix<float, Dynamic, Dynamic, RowMajor> imageRaw_1;
	imageRaw_1.resize(PointData_1.Num, 2);
	Matrix<float, Dynamic, Dynamic, RowMajor> imageRaw_2;
	imageRaw_2.resize(PointData_2.Num, 2);
	Matrix<float, K, 2, RowMajor> K_CenterPoint;
	Matrix<float, K, 1> az;

	float(*point_1)[M_Nano] = (float(*)[M_Nano]) PointData_1.pointData;
	for (int i = 0; i < PointData_1.Num; i++) {
		imageRaw_1(i, 0) = point_1[i][0];
		imageRaw_1(i, 1) = point_1[i][1];
	}

	float(*point_2)[M_Nano] = (float(*)[M_Nano]) PointData_2.pointData;
	for (int i = 0; i < PointData_2.Num; i++) {
		imageRaw_2(i, 0) = point_2[i][0];
		imageRaw_2(i, 1) = point_2[i][1];
	}

	Matrix<float, K, 1> vec;
	for (int cc = 0; cc < 300; cc++) {
		vec(cc) = cc + 1;
	}
	K_CenterPoint.col(0) = vec;
	K_CenterPoint.col(1) = vec;

	Matrix<float, Dynamic, 2> image_1;
	Matrix<float, Dynamic, 2> image_2;
	image_1 = imageRaw_1.rowwise() + T.col(0).transpose();
	image_2 = imageRaw_2.rowwise() + T.col(1).transpose();
	
	Matrix<float, K, 1> Q;
	Q = 0 * Q.array() + 0.50197*pow(10, -6);

	//h
	float h = 2 / Q.mean();
	float beta = 1.195200*pow(10,-8);

	//IterBegin
	for (int i = 0; i < maxNumIter; i++) {
		//������ŷ���������
		static const int Nu_1 = PointData_1.Num;
		Matrix<float, Dynamic, K, RowMajor> a_1,a_2;
		a_1.resize(PointData_1.Num, K);
		a_2.resize(PointData_2.Num, K);

		//������������ŷ������
		a_1 = image_1 * K_CenterPoint.transpose();
		a_2 = image_2 * K_CenterPoint.transpose();
		MatrixXf image_1_sqare = image_1.rowwise().squaredNorm();
		MatrixXf image_2_sqare = image_2.rowwise().squaredNorm();
		MatrixXf K_CenterPoint_sqare = K_CenterPoint.rowwise().squaredNorm();
		float *P_1, *C, *A_1;
		hipMalloc((void**)&P_1, PointData_1.Num*sizeof(float));
		hipMalloc((void**)&C, K*sizeof(float));
		hipMalloc((void**)&A_1, PointData_1.Num*K*sizeof(float));
		hipMemcpy(P_1, image_1_sqare.data(), PointData_1.Num * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(C, K_CenterPoint_sqare.data(), K * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(A_1, a_1.data(), PointData_1.Num*K * sizeof(float), hipMemcpyHostToDevice);
		dim3 Gridsize_1(PointData_1.Num/1024,1);
		dim3 Blocksize_1(1024);
		MatrixEularDis << <Gridsize_1, Blocksize_1 >> > (P_1, C, A_1, PointData_1.Num, K);
		float *P_2, *A_2;
		hipMalloc((void**)&P_2, PointData_2.Num * sizeof(float));
		hipMalloc((void**)&A_2, PointData_2.Num*K * sizeof(float));
		hipMemcpy(P_2, image_2_sqare.data(), PointData_2.Num * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(A_2, a_2.data(), PointData_2.Num*K * sizeof(float), hipMemcpyHostToDevice);
		dim3 Gridsize_2(PointData_2.Num / 1024, 1);
		dim3 Blocksize_2(1024);
		MatrixEularDis << <Gridsize_2, Blocksize_2 >> > (P_2, C, A_2, PointData_2.Num, K);
		hipMemcpy(a_1.data(), A_1, PointData_1.Num*K * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(a_2.data(), A_2, PointData_2.Num*K * sizeof(float), hipMemcpyDeviceToHost);

		//pk*S^-1.5*exp(-.5/S^2*||.||) and normalize
		for (int t = 0; t < PointData_1.Num; t++) {
			a_1.row(t) = (a_1.row(t).array()*Q.transpose().array().pow(2)*(-0.5)).array().exp();
			a_1.row(t) = a_1.row(t).array()*pk.transpose().array()*(Q.transpose().array().pow(1.5));
			a_1.row(t) = a_1.row(t) / (a_1.row(t).sum()+beta);
		}
		for (int t = 0; t < PointData_2.Num; t++) {
			a_2.row(t) = (a_2.row(t).array()*Q.transpose().array().pow(2)*(-0.5)).array().exp();
			a_2.row(t) = a_2.row(t).array()*pk.transpose().array()*(Q.transpose().array().pow(1.5));
			a_2.row(t) = a_2.row(t) / (a_2.row(t).sum() + beta);
		}

		Matrix<float, K, 2>lambda;
		Matrix<float, 2, K>W_1;
		Matrix<float, 2, K>W_2;
		Matrix<float, K, 2>b;
		Matrix<float, 2, 2>mW;
		Matrix<float, 2, 2>mX;
		Matrix<float, 1, 2>sumOfWeight;
		lambda.col(0) = a_1.colwise().sum();
		lambda.col(1) = a_2.colwise().sum();
		W_1.row(0) = (imageRaw_1.transpose()*a_1).row(0).cwiseProduct(Q.transpose());
		W_1.row(1) = (imageRaw_1.transpose()*a_1).row(1).cwiseProduct(Q.transpose());
		W_2.row(0) = (imageRaw_2.transpose()*a_2).row(0).cwiseProduct(Q.transpose());
		W_2.row(1) = (imageRaw_2.transpose()*a_2).row(1).cwiseProduct(Q.transpose());

		b.col(0) = lambda.col(0).cwiseProduct(Q);
		b.col(1) = lambda.col(1).cwiseProduct(Q);
		mW.col(0) = W_1.rowwise().sum();
		mW.col(1) = W_2.rowwise().sum();
		mX.col(0) = K_CenterPoint.transpose()*b.col(0);
		mX.col(1) = K_CenterPoint.transpose()*b.col(1);
		sumOfWeight(0, 0) = lambda.col(0).dot(Q);
		sumOfWeight(0, 1) = lambda.col(1).dot(Q);

		T.col(0) = (mX.col(0) - mW.col(0))/sumOfWeight(0,0);
		T.col(1) = (mX.col(1) - mW.col(1))/sumOfWeight(0,1);

		image_1 = imageRaw_1.rowwise() + T.col(0).transpose();
		image_2 = imageRaw_2.rowwise() + T.col(1).transpose();

		Matrix<float, 1, K>den;
		den = lambda.rowwise().sum().transpose();
		K_CenterPoint = a_1.transpose()*image_1 + a_2.transpose()*image_2;
		K_CenterPoint.col(0) = K_CenterPoint.col(0).cwiseQuotient(den.transpose());
		K_CenterPoint.col(1) = K_CenterPoint.col(1).cwiseQuotient(den.transpose());

		Matrix<float, K, 2>wnormes;
		Matrix<float, Dynamic, K> temp_a_1, temp_a_2;
		temp_a_1.resize(PointData_1.Num, K);
		temp_a_2.resize(PointData_2.Num, K);

		//������������ŷ������
		temp_a_1 = image_1 * K_CenterPoint.transpose();
		temp_a_2 = image_2 * K_CenterPoint.transpose();
		image_1_sqare = image_1.rowwise().squaredNorm();
		image_2_sqare = image_2.rowwise().squaredNorm();
		K_CenterPoint_sqare = K_CenterPoint.rowwise().squaredNorm();
		hipMemcpy(C, K_CenterPoint_sqare.data(), K * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(P_1, image_1_sqare.data(), PointData_1.Num * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(A_1, temp_a_1.data(), PointData_1.Num*K * sizeof(float), hipMemcpyHostToDevice);
		MatrixEularDis << <Gridsize_1, Blocksize_1 >> > (P_1, C, A_1, PointData_1.Num, K);
		hipMemcpy(P_2, image_2_sqare.data(), PointData_2.Num * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(A_2, temp_a_2.data(), PointData_2.Num*K * sizeof(float), hipMemcpyHostToDevice);
		MatrixEularDis << <Gridsize_2, Blocksize_2 >> > (P_2, C, A_2, PointData_2.Num, K);
		hipMemcpy(temp_a_1.data(), A_1, PointData_1.Num*K * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(temp_a_2.data(), A_2, PointData_2.Num*K * sizeof(float), hipMemcpyDeviceToHost);
		hipFree(P_1);
		hipFree(P_2);
		hipFree(C);
		hipFree(A_1);
		hipFree(A_2);

		wnormes.col(0) = (a_1.cwiseProduct(temp_a_1)).colwise().sum();
		wnormes.col(1) = (a_2.cwiseProduct(temp_a_2)).colwise().sum();

		Q = 3*den.transpose().cwiseQuotient(wnormes.rowwise().sum() + 3 * epsilon * den.transpose());

	}

	moveDistanceXY(0, 0) = -(T(0, 1) - T(0, 0));
	moveDistanceXY(0, 1) = -(T(1, 1) - T(1, 0));
	Tans = moveDistanceXY;
	return moveDistanceXY;
};
